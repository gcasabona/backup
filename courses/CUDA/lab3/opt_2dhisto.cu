#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"
#include "opt_2dhisto.h"
__global__ void opt_histo_kernel(uint32_t *input, uint8_t *resultopt, int width, int height);


void opt_2dhisto(int numBlocksW, int numBlocksH, int blockWidth, int blockHeight, uint32_t *inputd, uint8_t *resultd)
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */
	
	dim3 dimGrid(numBlocksW,numBlocksH);
	dim3 dimBlock(blockWidth, blockHeight);

	opt_histo_kernel<<<dimGrid, dimBlock>>>(inputd, resultd, INPUT_WIDTH, INPUT_HEIGHT);
	
}

uint32_t *AllocateDeviceMemData(){
	int size = INPUT_HEIGHT * INPUT_WIDTH * sizeof(uint32_t); 
	uint32_t *array_device ;
	hipMalloc((void**)&array_device, size);
	return array_device;
}

uint8_t *AllocateDeviceMemResult(){
	int size = HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint8_t);
	uint8_t *array_device;
	hipMalloc((void**)&array_device, size);
	return array_device;
}

void CopyToDevice(uint32_t *dataDevice, uint32_t **dataHost, uint8_t *resultDevice, uint8_t *resultHost){
	
	int result_size = HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint8_t);
	int data_size = INPUT_HEIGHT * INPUT_WIDTH * sizeof(uint32_t);
	for(int i=0; i < INPUT_HEIGHT; i++){
		hipMemcpy(dataDevice + i*INPUT_WIDTH, dataHost[i], data_size, hipMemcpyHostToDevice);
	}
	hipMemcpy(resultDevice, resultHost, result_size, hipMemcpyHostToDevice);

	
}

void CopyFromDevice(uint8_t *arrHost, uint8_t *arrDevice){
	
	int size = HISTO_WIDTH*HISTO_HEIGHT * sizeof(uint8_t);
	hipMemcpy(arrHost, arrDevice, size, hipMemcpyDeviceToHost);

}

void FreeCudaMemData(uint32_t *array){

	hipFree(array);
}

void FreeCudaMemResult(uint8_t *array){

	hipFree(array);
}

/* Include below the implementation of any other functions you need */
__global__ void opt_histo_kernel(uint32_t *input, uint8_t *resultopt, int width, int height){

	// This is an attempt of the block method: each threadblock calculates a sub-histogram in shared memory.

	// Step 1: Initialize partial histogram
	// Each thread:

	// Make thread & block IDs simple	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Find location of input element using rows and cols
	int row = by*blockDim.y + ty;
	int col = bx*blockDim.x + tx;

	// Value for each bin in sub-histogram
	float value = 0.0;

	if ((row < height) && (col < width)){
		value = input[row * width + col];
	//	resultopt[value]++;
	}

}

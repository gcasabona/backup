#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	const int tileWidth = 32;
	
	__shared__ float Msh[tileWidth][tileWidth];	
	__shared__ float Nsh[tileWidth][tileWidth];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	// Calculate the row index of the Pd element and M
	int Row = by*blockDim.y + ty; // yIndex
	
	// Calculate the column index of Pd and N
	int Col = bx*blockDim.x + tx; // xIndex
	
//	float Pvalue = 0;
	float Ppartial = 0;

        //printf("M.width = %d \n", M.width);
	// Read the input through shared memory: the point is to do the calculation in shared memory and write output to global memory. This is for COALESING.
	// To avoid bank conflicts, add PADDING
//	if (Row < P.height && Col < P.width){
		
		// Loop over tiles along M width and N height, which are the same
		int numTiles =  M.width/tileWidth;
		if ((M.width % tileWidth)>0){ numTiles++;   }
                 
		int overflow = (numTiles*tileWidth) - M.width;

		for (int m=0; m<numTiles; m++){

//		//FOR uncoalesced only! 
//		for (int k=0; k<M.width; k++) {
//			float Mel = M.elements[Row*M.width + k];
//              	float Nel = N.elements[k*N.width + Col];
//             		Pvalue += Mel * Nel;
//		}
		
//			Ppartial = 0;
			
//			Msh[ty][tx] = M.elements[Row*M.width + m*tileWidth + tx];
//			Nsh[ty][tx] = N.elements[(m*tileWidth + ty)*N.width + Col];
		 
//			__syncthreads();		
	
			
			if ((Row > M.height-1) || (m*tileWidth + tx > M.width-1)){
	 			Msh[ty][tx] = 0;
			}
			else {
				
			Msh[ty][tx] = M.elements[Row*M.width + m*tileWidth + tx];
			}
			if ((Col > N.width-1) || (m*tileWidth + ty > N.height-1)){
				Nsh[ty][tx] = 0;
			}
else {
			Nsh[ty][tx] = N.elements[(m*tileWidth + ty)*N.width + Col];
}
		 	//if (m == (numTiles - 1 )){
                        //	if (( tx  > (tileWidth - overflow) - 1 )|| (ty > (tileWidth - overflow) -1)){
                         //  		Msh[ty][tx] = 0;
                          //		Nsh[ty][tx] = 0;
			//	}
			//}
		        __syncthreads();
			
			for (int p=0; p<tileWidth; ++p){
				Ppartial += Msh[ty][p] * Nsh[p][tx];
                        //                         Pvalue += Ppartial;
			//	__syncthreads();

			}
                
                        __syncthreads();
        //		Pvalue += Ppartial;
		}
		//Pvalue += Ppartial;
//	}
	//__syncthreads();

	
	if (Row < P.height && Col < P.width){
		P.elements[Row * P.width + Col] = Ppartial;
	}
	
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_

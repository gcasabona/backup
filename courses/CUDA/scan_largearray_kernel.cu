#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels

#include <hip/hip_runtime.h>
#include <assert.h>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.

#define BLOCK_SIZE 256

//#define NUM_BANKS 16
//#define LOG_NUM_BANKS 4 
//#ifdef ZERO_BANK_CONFLICTS
//#define CONFLICT_FREE_OFFSET(n) \
 ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
//#else
//#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
//#endif
// Lab4: Host Helper Functions (allocate your own data structure...)

#endif

__global__ void scan(float *g_odata, float *g_idata, const int n);
void prescanArray(float *outArray, float *inArray, int numElements, float *sum);
__device__ void summation(float *sum);


// Lab4: Device Functions
__device__ void summation(float *sum){

	// This function should calculate the parallel prefix scan for the sum array
	//Now scan sum array

        int id = 0;
        int offsetSum = 1;
        for (int id = 0; id < gridDim.x; id++){ 
                for (int d = gridDim.x>>1; d > 0; d >>= 1) // build sum in place up the tree
                {
//                      __syncthreads();
                        if (id < d)
                        {
                                int a = offsetSum*(2*id+1)-1;
                                int b = offsetSum*(2*id+2)-1;
        
                                sum[b] += sum[a];
                        }
                offsetSum *= 2;
                }
        }
        sum[gridDim.x-1] = 0;
        
        for (int id = 0; id < gridDim.x; id++){
                for (int d = 1; d < gridDim.x; d *= 2) // traverse down tree & build scan
                {
                        offsetSum >>= 1;
                        __syncthreads();
                         if (id < d)
                        {
                                int a = offsetSum*(2*id+1)-1;
                                int b = offsetSum*(2*id+2)-1;
                                float t = sum[a];
                                sum[a] = sum[b];
                                sum[b] += t;
                        }
                }

        }
        
		


}

// Lab4: Kernel Functions
// n: block size. one thread can handle two elements
__global__ void scan(float *g_odata, float *g_idata, const int n, float *sum)
{

 	extern __shared__ float temp[]; // allocated on invocation: only needs to be as big as num threads in block

	int thid = threadIdx.x;	//thread id in block
	int gid = blockIdx.x*blockDim.x + thid;	//global id
	
	int offset = 1;
	
	//int numBlocks = dimGrid.x;

//	float sum[numBlocks];

	//Every thread handles two elements
	//temp[2*thid] = g_idata[2*gid]; // load input into shared memory
        //temp[2*thid+1] = g_idata[2*gid+1];

	
	temp[2*thid] = g_idata[2*thid]; // load input into shared memory
	temp[2*thid+1] = g_idata[2*thid+1];
	
/*
	int ai = thid;
	int bi = thid + (n/2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(ai);
	temp[ai + bankOffsetA] = g_idata[ai];
	temp[bi + bankOffsetB] = g_idata[bi]; 
*/

	// load input into shared memory.
 	// This is exclusive scan, so shift right by one and set first element to 0

	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
 	{
 		__syncthreads();
 		if (thid < d)
 		{
 			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
//			ai += CONFLICT_FREE_OFFSET(ai);
//			bi += CONFLICT_FREE_OFFSET(bi); 

			temp[bi] += temp[ai];
 		}
 		offset *= 2;
 	}
	

	// fill sum array with the sum of the first array
	__syncthreads();
	// Now we have the sum array. This needs to be added back to the blocks
	//temp[thid] += sum[blockIdx.x]; 		
	
	//if (thid==0) { temp[n – 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; }	
 	if (thid == 0) { temp[n - 1] = 0; } // clear the last element
 	


	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
        {
                offset >>= 1;
                __syncthreads();
                 if (thid < d)
                {
                        int ai = offset*(2*thid+1)-1;
                        int bi = offset*(2*thid+2)-1;
       
//			ai += CONFLICT_FREE_OFFSET(ai);
//			bi += CONFLICT_FREE_OFFSET(bi); 

	                 float t = temp[ai];
                        temp[ai] = temp[bi];
                        temp[bi] += t;
                }
        }

	//__syncthreads();
//	temp[thid] += sum[blockIdx.x];
	
 	__syncthreads();
 //	g_odata[2*gid] = temp[2*thid]; // write results to device memory
 //	g_odata[2*gid+1] = temp[2*thid+1];
	//g_odata[ai] = temp[ai + bankOffsetA];
	//g_odata[bi] = temp[bi + bankOffsetB]; 						
 	g_odata[2*thid] = temp[2*thid]; // write results to device memory
 	g_odata[2*thid+1] = temp[2*thid+1];
}


	
	

// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{
	// Divide input array into blocks
	// Remember that each thread can handle two elements
	// BLOCK_SIZE is set above as a constant
	const int numBlocks = ceil(numElements/BLOCK_SIZE);

	float *sum;
	float *inc;
	
	// Allocate global device memory for arrays to communicate sum data
	hipMalloc((void**)&sum, numBlocks*sizeof(float));
	hipMalloc((void**)&inc, numBlocks*sizeof(float));

	
	dim3 dimGrid(1);
	dim3 dimBlock(numElements);
    	//scan<<<dimGrid, dimBlock, 2*sizeof(float)*numElements+1>>>(outarray, inArray, numElements);
	scan<<<dimGrid, dimBlock, 2*sizeof(float)*numElements+1>>>(outArray, inArray, numElements, sum); 
	
//	cudaThreadSynchronize();

	//summation<<< >>>();
//	cudaThreadSynchronize();    
}
// **===-----------------------------------------------------------===**


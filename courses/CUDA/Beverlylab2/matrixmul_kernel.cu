#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#define TILE_WIDTH 8

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	const int tileWidth = 8;	
	//__shared__ float Msh[TILE_WIDTH * TILE_WIDTH];
	//__shared__ float Nsh[TILE_WIDTH * TILE_WIDTH];
	__shared__ float Msd[tileWidth * tileWidth];	
	__shared__ float Nsh[tileWidth * tileWidth];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	// Calculate the row index of the Pd element and M
	int Row = by*blockDim.y + ty; // yIndex
	
	// Calculate the column index of Pd and N
	int Col = bx*blockDim.x + tx; // xIndex
	
	float Pvalue = 0;


	// Read the input through shared memory: the point is to do the calculation in shared memory and write output to global memory. This is for COALESING.
	// To avoid bank conflicts, add PADDING
	if (Row < P.height && Col < P.width){

		for (int k=0; k<M.width; k++) {
			float Mel = M.elements[Row*M.width + k];
                	float Nel = N.elements[k*N.width + Col];
       	        	Pvalue += Mel * Nel;
				//Pvalue += Md[ty*P.width + k] * Nd[k*P.height + Col];
		}
	__syncthreads();
	}
	// Loop over tiles. 
//	for (int m = 0; m < gridDim.x; ++m) {
//		// Coolaborative loading of d_M and d_N tiles into shared memory
//		Md[ty][tx] = M.elements[Row*P.width + m*TILE_WIDTH + tx];
//		Nd[ty][tx] = N.elements[(m*TILE_WIDTH + ty)*P.width + Col];
//		__syncthreads();
	
//		for (int k = 0; k < TILE_WIDTH; ++k) {
//			Pvalue += Md[ty][k] * Nd[k][tx];
//		}
//		__syncthreads();
//	}
	// each thread computes one element
	// of the block sub-matrix

	//for (int k = 0; k < P.width; ++k) {
	//	float Mel = M.elements[Row * P.width + k];
	//	float Nel = N.elements[k * P.width + Col];
	//	Pvalue += Mel * Nel;
	//}
	
	//s_data[Row * P.width + Col] = Pvalue;
	if (Row < P.height && Col < P.width){
		P.elements[Row * P.width + Col] = Pvalue;
	}
	// Write the result through SMEM --> NOT RIGHT YET
	//if(Col < P.width && Row < P.height){
	//	P.elements[WHAT] = s_data[WHAT];
	//}
	
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
